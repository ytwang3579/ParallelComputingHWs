#include "hip/hip_runtime.h"
//***********************************************************************************
// 2018.04.01 created by Zexlus1126
//
//    Example 002
// This is a simple demonstration on calculating merkle root from merkle branch 
// and solving a block (#286819) which the information is downloaded from Block Explorer 
//***********************************************************************************

#include <iostream>
#include <fstream>
#include <string>

#include <cstdio>
#include <cstring>

#include <cassert>

#include "sha256.h"

////////////////////////   Block   /////////////////////

typedef struct _block
{
    unsigned int version;
    unsigned char prevhash[32];
    unsigned char merkle_root[32];
    unsigned int ntime;
    unsigned int nbits;
    unsigned int nonce;
}HashBlock;


////////////////////////   Utils   ///////////////////////

//convert one hex-codec char to binary
__host__ __device__  unsigned char decode(unsigned char c)
{
    switch(c)
    {
        case 'a':
            return 0x0a;
        case 'b':
            return 0x0b;
        case 'c':
            return 0x0c;
        case 'd':
            return 0x0d;
        case 'e':
            return 0x0e;
        case 'f':
            return 0x0f;
        default:
            return c-'0';
    }

}


// convert hex string to binary
//
// in: input string
// string_len: the length of the input string
//      '\0' is not included in string_len!!!
// out: output bytes array
__host__ __device__ void convert_string_to_little_endian_bytes(unsigned char* out, char *in, size_t string_len, int idx)
{
    // if(idx == 0) printf("In: %s\n", in);
    assert(string_len % 2 == 0);

    size_t s = 0;
    size_t b = string_len/2-1;
    // if(idx == 0) printf("Out:");
    for(s, b; s < string_len; s+=2, --b)
    {   // if(idx == 0) printf("%x %x -->", decode(in[s]), decode(in[s+1]));
        out[b] = (unsigned char)(decode(in[s])<<4) + decode(in[s+1]);
        // if(idx == 0) printf("%02x\n", out[b]);
    }
    // if(idx == 0) printf("\n");
}

// print out binary array (from highest value) in the hex format
__host__ __device__ void print_hex(unsigned char* hex, size_t len)
{
    for(int i=0;i<len;++i)
    {
        printf("%02x", hex[i]);
    }
}


// print out binar array (from lowest value) in the hex format
__host__ __device__ void print_hex_inverse(unsigned char* hex, size_t len)
{
    for(int i=len-1;i>=0;--i)
    {
        printf("%02x", hex[i]);
    }
}

__host__ __device__ int little_endian_bit_comparison(const unsigned char *a, const unsigned char *b, size_t byte_len)
{
    // compared from lowest bit
    for(int i=byte_len-1;i>=0;--i)
    {
        if(a[i] < b[i])
            return -1;
        else if(a[i] > b[i])
            return 1;
    }
    return 0;
}

void getline(char *str, size_t len, FILE *fp)
{

    int i=0;
    while( i<len && (str[i] = fgetc(fp)) != EOF && str[i++] != '\n');
    str[len-1] = '\0';
}

////////////////////////   Hash   ///////////////////////

__host__ __device__  void double_sha256(SHA256 *sha256_ctx, unsigned char *bytes, size_t len)
{
    SHA256 tmp;
    sha256(&tmp, (BYTE*)bytes, len);
    sha256(sha256_ctx, (BYTE*)&tmp, sizeof(tmp));
}


////////////////////   Merkle Root   /////////////////////


// calculate merkle root from several merkle branches
// root: output hash will store here (little-endian)
// branch: merkle branch  (big-endian)
// count: total number of merkle branch
void calc_merkle_root(unsigned char *root, int count, char **branch)
{
    size_t total_count = count; // merkle branch
    unsigned char *raw_list = new unsigned char[(total_count+1)*32];
    unsigned char **list = new unsigned char*[total_count+1];

    // copy each branch to the list
    for(int i=0;i<total_count; ++i)
    {
        list[i] = raw_list + i * 32;
        //convert hex string to bytes array and store them into the list
        convert_string_to_little_endian_bytes(list[i], branch[i], 64, 1);
    }

    list[total_count] = raw_list + total_count*32;


    // calculate merkle root
    while(total_count > 1)
    {
        
        // hash each pair
        int i, j;

        if(total_count % 2 == 1)  //odd, 
        {
            memcpy(list[total_count], list[total_count-1], 32);
        }

        for(i=0, j=0;i<total_count;i+=2, ++j)
        {
            // this part is slightly tricky,
            //   because of the implementation of the double_sha256,
            //   we can avoid the memory begin overwritten during our sha256d calculation
            // double_sha:
            //     tmp = hash(list[0]+list[1])
            //     list[0] = hash(tmp)
            double_sha256((SHA256*)list[j], list[i], 64);
        }

        total_count = j;
    }

    memcpy(root, list[0], 32);

    delete[] raw_list;
    delete[] list;
}

__device__ bool flag = true;
__global__ void clearflag() { flag = true; }
__global__ void solve(char* version, 
    char* prevhash,
    char* ntime,
    char* nbits,
    unsigned char* merkle_root,
    int& tx,
    unsigned int& ans)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = gridDim.x * blockDim.x;

    // for(int i=idx; i<tx; i+=gridStride) {
    //     merkle_branch[i] = raw_merkle_branch + i * 65;
    //     merkle_branch[i][64] = '\0';
    // }

    // __syncthreads();

    // **** calculate merkle root ****

    
    // if(threadIdx.x == 0) {
        
    // }

    // __syncthreads();


    // **** solve block ****
    if(idx == 0) {
        printf("Block info (big): \n");
        printf("  version:  %s\n", version);
        printf("  pervhash: %s\n", prevhash);
        printf("  merkleroot: "); print_hex_inverse(merkle_root, 32); printf("\n");
        printf("  nbits:    %s\n", nbits);
        printf("  ntime:    %s\n", ntime);
        printf("  nonce:    ???\n\n");
    }
    

    HashBlock block;

    // convert to byte array in little-endian
    convert_string_to_little_endian_bytes((unsigned char *)&block.version, version, 8, idx);
    convert_string_to_little_endian_bytes(block.prevhash,                  prevhash,    64, idx);
    memcpy(block.merkle_root, merkle_root, 32);
    convert_string_to_little_endian_bytes((unsigned char *)&block.nbits,   nbits,     8, idx);
    convert_string_to_little_endian_bytes((unsigned char *)&block.ntime,   ntime,     8, idx);
    block.nonce = 0;

    // if(idx == 0) {
    //     printf("Block info (big): \n");
    //     printf("  version:  %x\n", block.version);
    //     printf("  pervhash: %x\n", block.prevhash);
    //     printf("  merkleroot: "); print_hex_inverse(block.merkle_root, 32); printf("\n");
    //     printf("  nbits:    %x\n", block.nbits);
    //     printf("  ntime:    %x\n", block.ntime);
    //     printf("  nonce:    ???\n\n");
    // }
    
    
    // ********** calculate target value *********
    // calculate target value from encoded difficulty which is encoded on "nbits"
    unsigned int exp = block.nbits >> 24;
    unsigned int mant = block.nbits & 0xffffff;
    unsigned char target_hex[32] = {};
    
    unsigned int shift = 8 * (exp - 3);
    unsigned int sb = shift / 8;
    unsigned int rb = shift % 8;
    
    // little-endian
    target_hex[sb    ] = (mant << rb);
    target_hex[sb + 1] = (mant >> (8-rb));
    target_hex[sb + 2] = (mant >> (16-rb));
    target_hex[sb + 3] = (mant >> (24-rb));
    
    if(idx == 0) {
        printf("Target value (big): ");
        print_hex_inverse(target_hex, 32);
        printf("\n"); 
    }
    


    // ********** find nonce **************
    
    SHA256 sha256_ctx;
    
    for(block.nonce=idx; block.nonce<=0xffffffff;block.nonce += gridStride)
    {   
        if(flag == false) break;

        //sha256d
        double_sha256(&sha256_ctx, (unsigned char*)&block, sizeof(block));

        if(block.nonce % 1000000 == 0)
        {
            // printf("hash #%10u (big): ", block.nonce);
            // print_hex_inverse(sha256_ctx.b, 32);
            // printf("\n");
        }

        if(little_endian_bit_comparison(sha256_ctx.b, target_hex, 32) < 0)  // sha256_ctx < target_hex
        {
            printf("Found Solution!!\n");
            printf("hash #%10u (big): ", block.nonce);
            print_hex_inverse(sha256_ctx.b, 32);
            printf("\n\n");
            ans = block.nonce;
            flag = false;
        }

    }


    // print result

    // //little-endian
    // printf("hash(little): ");
    // print_hex(sha256_ctx.b, 32);
    // printf("\n");

    // //big-endian
    // printf("hash(big):    ");
    // print_hex_inverse(sha256_ctx.b, 32);
    // printf("\n\n");



}

int main(int argc, char **argv)
{
    fprintf(stderr, "Hello world!\n");
    if (argc != 3) {
        fprintf(stderr, "usage: cuda_miner <in> <out>\n");
    }
    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");

    int totalblock;

    fprintf(stderr, "Getting device props...\n");
    int deviceID;
    hipGetDevice(&deviceID);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceID);

    int Block_Per_Grid = props.multiProcessorCount * props.warpSize;
    int Thread_Per_Block = props.maxThreadsPerMultiProcessor / props.warpSize;
    fprintf(stderr, "%d %d %d\n", Block_Per_Grid, props.maxThreadsPerMultiProcessor, Thread_Per_Block);


    fscanf(fin, "%d\n", &totalblock);
    fprintf(fout, "%d\n", totalblock);

    for(int i=0;i<totalblock;++i)
    {
        
        // **** read data *****
        char version[9];
        char prevhash[65];
        char ntime[9];
        char nbits[9];
        int tx;
        char *raw_merkle_branch;
        char **merkle_branch;

        char *version_cuda, *prevhash_cuda, *ntime_cuda, *nbits_cuda, *raw_merkle_branch_cuda;
        int *tx_cuda;
        char** merkle_branch_cuda;
        
        hipMalloc(&version_cuda, sizeof(char) * 9);
        hipMalloc(&prevhash_cuda, sizeof(char) * 65);
        hipMalloc(&ntime_cuda, sizeof(char) * 9);
        hipMalloc(&nbits_cuda, sizeof(char) * 9);
        hipMalloc(&tx_cuda, sizeof(int));

        getline(version, 9, fin);
        getline(prevhash, 65, fin);
        getline(ntime, 9, fin);
        getline(nbits, 9, fin);
        fscanf(fin, "%d\n", &tx);

        hipMemcpyAsync(version_cuda, version, 9, hipMemcpyHostToDevice);
        hipMemcpyAsync(prevhash_cuda, prevhash, 65, hipMemcpyHostToDevice);
        hipMemcpyAsync(ntime_cuda, ntime, 9, hipMemcpyHostToDevice);
        hipMemcpyAsync(nbits_cuda, nbits, 9, hipMemcpyHostToDevice);
        hipMemcpyAsync(tx_cuda, &tx, 1, hipMemcpyHostToDevice);

        hipHostMalloc(&raw_merkle_branch, tx*65*sizeof(char));
        hipHostMalloc(&merkle_branch, tx*sizeof(char*));

        hipMalloc(&raw_merkle_branch_cuda, sizeof(char) * tx*65);
        hipMalloc(&merkle_branch_cuda, sizeof(char*) * tx);
        
        fprintf(stderr, "Building environment for merkle root...\n");
        
        #pragma omp parallel for scheduled(static)
        for(int i=0;i<tx;++i)
        {
            merkle_branch[i] = raw_merkle_branch + i * 65;
            merkle_branch[i][64] = '\0';
        }
        for(int i=0;i<tx;++i) getline(merkle_branch[i], 65, fin);

        
        // hipMemcpyAsync(raw_merkle_branch_cuda, raw_merkle_branch, tx*65, hipMemcpyHostToDevice);
        
        fprintf(stderr, "Calculate merkle root\n");
        unsigned char *merkle_root, *merkle_root_cuda;
        hipHostMalloc(&merkle_root, 32*sizeof(unsigned char));
        hipMalloc(&merkle_root_cuda, 32*sizeof(unsigned char));

        calc_merkle_root(merkle_root, tx, merkle_branch);
        hipMemcpyAsync(merkle_root_cuda, merkle_root, 32*sizeof(unsigned char), hipMemcpyHostToDevice);

        printf("merkle root(little): ");
        print_hex(merkle_root, 32);
        printf("\n");

        printf("merkle root(big):    ");
        print_hex_inverse(merkle_root, 32);
        printf("\n");

        fprintf(stderr, "Entering solve...\n");
        unsigned int *ans, *ans_cuda;
        
        hipMalloc(&ans_cuda, sizeof(unsigned int));

        solve<<<Block_Per_Grid, Thread_Per_Block>>>(version_cuda, prevhash_cuda, ntime_cuda, nbits_cuda,
            merkle_root_cuda, *tx_cuda, *ans_cuda);

        hipHostMalloc(&ans, sizeof(unsigned int));
        
        hipMemcpy(ans, ans_cuda, sizeof(unsigned int), hipMemcpyDeviceToHost);

        for(int i=0;i<4;++i)
        {
            fprintf(fout, "%02x", ((unsigned char*)ans)[i]);
        }
        fprintf(fout, "\n"); 

        fprintf(stderr, "Finish solve...\n");

        clearflag<<<1,1>>>();
        hipDeviceSynchronize();

        hipFree(version_cuda);
        hipFree(prevhash_cuda);
        hipFree(ntime_cuda);
        hipFree(nbits_cuda);
        hipFree(tx_cuda);
        hipHostFree(raw_merkle_branch);
        hipHostFree(merkle_branch);

        hipFree(raw_merkle_branch_cuda);
        hipFree(merkle_branch_cuda);
        hipHostFree(merkle_root_cuda);
        hipFree(merkle_root_cuda);

        hipFree(ans_cuda);
        hipHostFree(ans);
    }

    return 0;
}

